#include <iostream>
#include "batch_matmul.cuh"

#define EQUAL

#ifdef EQUAL
  // equal is when two tensor has the batches of matrices
  // let's say 
  // Tensor A has         shape[3, 4, M, K]
  // Tensor B has         shape[3, 4, K, N]
  // Tensor C will output shape[3, 4, M, N]
#else
  // this is when one tensor has an arbitary shape
  // and the other is a matrix of 2 dimension 
  // let's say
  // Tensor A has         shape[3, 4, M, K]
  // Tensor B has         shape      [K, M]
  // Tensor C will output shape[3, 4, M, N] 
#endif

// put your param here
#define BATCH_SIZE 3*4
#define M 1024
#define N 1024
#define K 1024

#define MIN -2
#define MAX 2

#define RANGE (MAX + 1 - MIN) + MAX

int main(){

  size_t A_size = BATCH_SIZE*M*K;
#ifdef EQUAL
  std::cout << "Doing elementwise matmul" << std::endl;
  size_t B_size = BATCH_SIZE*K*N;
#else
  std::cout << "Doing broadcasted matmul" << std::endl;
  size_t B_size = K*N;
#endif
  size_t C_size = BATCH_SIZE*M*N; 

  float* A = new float[A_size];
  float* B = new float[B_size];
  float* C = new float[C_size];

  float* A_dev = nullptr;
  float* B_dev = nullptr;
  float* C_dev = nullptr; 

  // fill in a random number from range MIN to MAX
  for (int i = 0; i < A_size; i++){
    A[i] = rand() % RANGE;
  }

  for (int i = 0; i< B_size; i++){
    B[i] = rand() % RANGE;
  }
  
  hipMalloc((void**)&A_dev, sizeof(float)*A_size);
  hipMalloc((void**)&B_dev, sizeof(float)*B_size);
  hipMalloc((void**)&C_dev, sizeof(float)*C_size);

  hipMemcpy((void*)A_dev, A, sizeof(float)*A_size, hipMemcpyHostToDevice);
  hipMemcpy((void*)B_dev, B, sizeof(float)*B_size, hipMemcpyHostToDevice);
  hipMemcpy((void*)C_dev, C, sizeof(float)*C_size, hipMemcpyHostToDevice);

  std::cout << "CPU ops" << std::endl;
  elementwise_gemm_cpu(A, B, C, BATCH_SIZE, M, N, K);
  std::cout << "GPU ops" << std::endl;
  elementwise_gemm(A_dev, B_dev, C_dev, BATCH_SIZE, M, N, K);


  delete[] A;
  delete[] B;
  delete[] C;

  hipFree(A_dev);
  hipFree(B_dev);
  hipFree(C_dev);

  return 0;
}